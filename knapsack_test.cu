#include <iostream>
#include <fstream>
#include <stdio.h>
#include <vector>
#include <string>

#include "knapsack.h"
#include "testing.h"

bool verbose = false;

void usage() {
  std::cerr << "Usage: ./executable <input file> <budget> <option>\n";
}

bool load_stocks(std::string filename, std::vector<Stock>& stocks) {
  std::ifstream inFile(filename);

  if(!inFile.is_open()) {
    std::cerr << "Could not open file " << filename << std::endl;
    return false;
  }

  int id, price, quantity;
  std::vector< std::pair<int, int> > distr;

  while(inFile >> id) {
    inFile >> price >> quantity;
    int a, b;

    inFile >> a;
    inFile >> a;

    while(a != -1888) {
      inFile >> b;

      distr.push_back({a, b});

      inFile >> a;
    }

    stocks.push_back(Stock(id, price, quantity, distr));
    distr.clear();
  }

  inFile.close();

  return true;
}

int main(int argc, char** argv) {
  if(argc != 3 && argc != 4) {
    usage();
    return 1;
  }

  if(argc == 4 && std::string(argv[3]) == "-v") {
    verbose = true;
  }

  std::string filename(argv[1]);
  int budget = atoi(argv[2]);

  if(budget) {
    if(budget < 0) {
      std::cerr << "Budget cannot be negative\n";
      return 1;
    }
    std::cout << "Budget is " << budget << std::endl;

    if(budget > 4000000) {
      std::cerr << "Budget cannot be over 1 million\n";
      return 1;
    }
  } else {
    std::cout << "Budget is zero; setting to $1,000,000\n";
    budget = 1000000;
  }

  std::vector<Stock> stocks;
  bool load_success = load_stocks(filename, stocks);
  if(!load_success) {
    return 1;
  }
  
  printf("Num items: %lu\n", stocks.size());
  int v;
  int* stock_values = new int[stocks.size()];
  for(size_t i = 0; i < stocks.size(); ++i) {
    stock_values[i] = stocks[i].expectedValue();
  }
  std::vector< std::pair<int, int> > solution;
  knapsack(stocks, stock_values, solution, v, stocks.size(), budget);
  std::cout << v << std::endl;

  free(stock_values);

  return 0;
}