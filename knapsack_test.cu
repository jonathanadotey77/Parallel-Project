#include <iostream>
#include <fstream>
#include <stdio.h>
#include <vector>
#include <string>

#include "knapsack.h"
#include "testing.h"

bool verbose = false;

void usage() {
  std::cerr << "Usage: ./executable <input file> <budget> <option>\n";
}

int main(int argc, char** argv) {
  if(argc != 3 && argc != 4) {
    usage();
    return 1;
  }

  if(argc == 4 && std::string(argv[3]) == "-v") {
    verbose = true;
  }

  std::string filename(argv[1]);
  int budget = atoi(argv[2]);

  if(budget) {
    if(budget < 0) {
      std::cerr << "Budget cannot be negative\n";
      return 1;
    }
    std::cout << "Budget is " << budget << std::endl;

    if(budget > 4000000) {
      std::cerr << "Budget cannot be over 1 million\n";
      return 1;
    }
  } else {
    std::cout << "Budget is zero; setting to $1,000,000\n";
    budget = 1000000;
  }

  std::vector<Stock> stocks;
  bool load_success = load_stocks(filename, stocks);
  if(!load_success) {
    return 1;
  }
  
  printf("Num items: %lu\n", stocks.size());
  int v;
  std::vector< std::pair<int, int> > solution;
  knapsack(stocks, solution, v, stocks.size(), budget);
  std::cout << v << std::endl;

  return 0;
}