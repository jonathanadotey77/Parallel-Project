#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <random>
#include <ctime>

#include "knapsack.h"
#include "timer.h"
#include "stock.h"

extern bool verbose;

namespace cg = cooperative_groups;

void allocateItems(const std::vector<Stock>& stocks,
  const int* stock_values,
  int*& item_costs, int*& item_values, int*& item_quantities,
  unsigned short*& chosen) {
  
  hipMallocManaged(&item_costs, stocks.size() * sizeof(int));
  hipMallocManaged(&item_values, stocks.size() * sizeof(int));
  hipMallocManaged(&item_quantities, stocks.size() * sizeof(int));
  hipMallocManaged(&chosen, stocks.size() * sizeof(unsigned short));
  for(size_t i = 0; i < stocks.size(); ++i) {
    int w = stocks[i].getPrice();
    int v = stock_values[i];
    int q = stocks[i].getQuantity();
    item_costs[i] = w;
    item_values[i] = v;
    item_quantities[i] = q;
    chosen[i] = 0;
  }
}

bool allocateTables(int*& table, int*& pointers, int*& quants,
  const size_t num_items, const int budget) {
  const size_t size = (size_t)((num_items+1) * ((unsigned long)budget + 1));

  hipError_t err1 = hipMalloc(&table, size * sizeof(int));
  if(err1 != hipSuccess) {
    printf("Table alloc failed\n");
    return false;
  }
  hipError_t err2 = hipMallocManaged(&pointers, size * sizeof(int));
  if(err2 != hipSuccess) {
    printf("Pointer alloc failed\n");
    hipFree(table);
    return false;
  }
 
  hipError_t err3 = hipMallocManaged(&quants, size * sizeof(int));
  if(err3 != hipSuccess) {
    printf("Pointer alloc failed\n");
    hipFree(table);
    hipFree(pointers);
    return false;
  }

  if(verbose) {
    printf("Total Table memory: %lu\n", 3 * size * sizeof(int));
  }

  return false;
}

void freeTable(int* table, int* pointers, int* quants) {
  hipFree(table);
  hipFree(pointers);
  hipFree(quants);
}

void freeItems(int* item_costs, int* item_values, int* item_quantities,
  unsigned short* chosen) {

  hipFree(chosen);
  hipFree(item_costs);
  hipFree(item_values);
  hipFree(item_quantities);
}

__global__ void knapsackKernel(
  const int* item_costs,
  const int* item_values,
  const int* item_quantities,
  int* table,
  int* pointers,
  int* quants,
  const size_t num_items,
  const int offset,
  const int budget) {

  cg::grid_group grid = cg::this_grid();
  const int w = blockIdx.x * blockDim.x + threadIdx.x + offset;
  const int cols = budget+1;

  if(w <= num_items) {
    table[w * cols + 0] = 0;
    pointers[w * cols + 0] = 0;
    quants[w * cols + 0] = 0;
  }

  if(w <= budget) {
    table[w] = 0;
    pointers[w] = 0;
    quants[w] = 0;
  }
  
  grid.sync();

  for(int i = 1; i <= num_items; ++i) {
    if(w <= budget) {
      int a = table[(i-1)*cols + w];
      int cost = item_costs[i-1];
      int value = item_values[i-1];
      int quantity = item_quantities[i-1];
      int last = w;
      int quant = 0;
      
      if(value > 0) {
        for(int q = 1; q <= quantity && q * cost <= w; ++q) {
          int val = table[(i-1)*cols + (w - q * cost)] + (q * value);
          if(val > a) {
            a = val;
            last = (w - q * cost);
            quant = q;
          }
        }
      }

      table[i*cols + w] = a;
      pointers[i*cols + w] = last;
      quants[i*cols + w] = quant;
    }
    grid.sync();
  }
}

__global__ void pullValue(int* table, int* val, size_t idx) {
  if(blockIdx.x * blockDim.x * threadIdx.x == 0) {
    *val = table[idx];
  }
}

__global__ void getChosenStocks(int* table, int* pointers, int* quants,
  int* item_costs, int* item_values, int* item_quantities,
  unsigned short* chosen, int num_items, int budget, int result) {

  if(blockIdx.x + threadIdx.x == 0) {
    int cols = budget+1;
    int w = budget;
    for(int i = num_items; i > 0 && result > 0; --i) {
      chosen[i-1] = quants[i*cols + w];
      w = pointers[i*cols + w];
      result = table[(i-1)*cols] + w;
    }
  }
}

void knapsack(const std::vector<Stock>& stocks,
  const int* stock_values,
  std::vector< std::vector<int> >& solution,
  int& total, size_t num_items, int budget, double* timer) {

  int* val;
  unsigned short* chosen;
  int* table;
  int* pointers;
  int* quants;
  int offset = 0;

  assert(num_items == stocks.size());

  hipMallocManaged(&val, 1 * sizeof(int));
  *val = 0;

  int* item_costs;
  int* item_values;
  int* item_quantities;

  //Allocate dedicated and shared memory for GPU usage
  allocateItems(stocks, stock_values, item_costs, item_values, item_quantities, chosen);
  allocateTables(table, pointers, quants, num_items, budget);

  //Args for knapsack kernel
  void* args[] = {
    &item_costs,
    &item_values,
    &item_quantities,
    &table,
    &pointers,
    &quants,
    &num_items,
    &offset,
    &budget
  };  

  //Dimensions for kernel
  const unsigned int max_blocks = 120;
  const unsigned int work_per_call = max_blocks * 1024;

  //We may not be able to run knapsack on the whole table at once
  //So, we run on "strips" of the table, which achieves the same result
  auto start = clock_time();
  for(int i = 0; i < budget; i += work_per_call) {
    offset = i;
    if(verbose) {
      printf("work done so far: %7d | launching %d total threads\n", i, work_per_call);
    }
    dim3 dimGrid(max_blocks, 1, 1);
    dim3 dimBlock(1024, 1, 1);
    //Launch cooperative kernel to ensure that all threads can be synchronized together
    hipLaunchCooperativeKernel((void*)knapsackKernel, dimGrid, dimBlock, args);
  }
  pullValue<<< 1, 1 >>>(table, val, num_items*(budget+1) + budget);
  hipDeviceSynchronize();

  int v = *val;
  if(verbose) {
    printf("Total value is %d, fetching solution\n", v);
  }

  //The "backtrack" step of knapsack
  //Since the table is in GPU dedicated memory, we launch a kernel with thread
  getChosenStocks<<< 1, 1 >>>(table, pointers, quants, item_costs, item_values, item_quantities,
          chosen, num_items, budget, v);
  hipDeviceSynchronize();
  auto end = clock_time();
  if(timer != NULL) *timer = calc_time(start, end);
  if(verbose) {
    printf("Fetched solution\n");
  }

  int total_weight = 0;
  int total_value = 0;

  //Move solution into a vector
  solution.clear();
  for(int i = 0; i < num_items; ++i) {
    if(chosen[i] == 0) {
      continue;
    }

    solution.push_back({stocks[i].getID(), item_costs[i], chosen[i]});

    total_weight += chosen[i] * item_costs[i];
    total_value += chosen[i] * item_values[i];
    if(verbose) {
      printf("Chose %d of stock #%d\n", chosen[i], i+1);
    }
  }

  if(total_weight > budget || total_value != v) {
    printf("Error with knapsack\n");
  }

  //Free memory
  freeItems(item_costs, item_values, item_quantities, chosen);
  freeTable(table, pointers, quants);
  hipFree(val);

  total = v;
}

bool mapRankToGPU(int myrank) {
  int cudaDeviceCount, cE;

  if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess ) {
    printf(" Unable to determine cuda device count, error is %d, count is %d\n", cE, cudaDeviceCount );
    return false;
  }

  if( (cE = hipSetDevice( myrank % cudaDeviceCount )) != hipSuccess ) {
    printf(" Unable to have gpu rank %d set to cuda device %d, error is %d \n", myrank, (myrank % cudaDeviceCount), cE);
    return false;
  }

  if(verbose) {
    printf("Mapping gpu rank %d to CUDA device %d\n", myrank, (myrank % cudaDeviceCount));
  }

  return true;
}