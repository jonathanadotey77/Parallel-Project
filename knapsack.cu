#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <vector>
#include <iostream>
#include <random>
#include <ctime>

#include "stock.h"

namespace cg = cooperative_groups;


__device__ unsigned int knapsackBarrier = 0;

// addr must be aligned properly.
__device__ unsigned int atomicLoad(const unsigned int *addr)
{
  const volatile unsigned int *vaddr = addr; // volatile to bypass cache
  __threadfence(); // for seq_cst loads. Remove for acquire semantics.
  const unsigned int value = *vaddr;
  // fence to ensure that dependent reads are correctly ordered
  __threadfence(); 
  return value; 
}

// addr must be aligned properly.
__device__ void atomicStore(unsigned int *addr, unsigned int value)
{
  volatile unsigned int *vaddr = addr; // volatile to bypass cache
  // fence to ensure that previous non-atomic stores are visible to other threads
  __threadfence(); 
  *vaddr = value;
}

void allocateItems(const std::vector<Stock>& stocks,
                    int*& item_costs, int*& item_values, int*& item_quantities,
                    unsigned short*& chosen) {
  
  hipMallocManaged(&item_costs, stocks.size() * sizeof(int));
  hipMallocManaged(&item_values, stocks.size() * sizeof(int));
  hipMallocManaged(&item_quantities, stocks.size() * sizeof(int));
  hipMallocManaged(&chosen, stocks.size() * sizeof(unsigned short));
  for(size_t i = 0; i < stocks.size(); ++i) {
    int w = stocks[i].getPrice();
    int v = stocks[i].expectedValue();
    int q = stocks[i].getQuantity();
    item_costs[i] = w;
    item_values[i] = v;
    item_quantities[i] = q;
    chosen[i] = 0;
  }
}

void allocateTable(int*& table, const size_t num_items, const int budget) {
  hipMalloc(&table, (num_items+1) * (budget + 1) * sizeof(int));
}

void freeTable(int* table) {
  hipFree(table);
}

void freeItems(int* item_costs, int* item_values, int* item_quantities,
               unsigned short* chosen) {
  hipFree(chosen);
  hipFree(item_costs);
  hipFree(item_values);
  hipFree(item_quantities);
}

__global__ void knapsackKernel(int* item_costs,
                         int* item_values,
                         int* item_quantities,
                         unsigned short* chosen,
                         int* table,
                         size_t num_items,
                         int budget) {

  cg::grid_group grid = cg::this_grid();
  const int w = blockIdx.x * blockDim.x + threadIdx.x;

  
  const int rows = num_items+1;
  const int cols = budget+1;
  //table[item][cost] = table[item*cols + cost];
  if(w <= num_items) {
    table[w * cols + 0] = 0;
  }

  // __syncthreads();

  if(w <= budget) {
    table[w] = 0;
  }
  
  // __syncthreads();
  grid.sync();

  if(w <= budget) {
    for(int i = 1; i <= num_items; ++i) {
      table[i*cols + w] = table[(i-1)*cols + w];
      int a = table[i*cols + w];
      for(int q = 1; q <= item_quantities[i-1] && q * item_costs[i-1] <= w; ++q) {
        int val = table[(i-1)*cols + (w - q * item_costs[i-1])] + (q * item_values[i-1]);
        if(val > a) {
          a = val;
        }
      }
      table[i*cols + w] = a;
      // __syncthreads();
      grid.sync();
    }
  }
}

__global__ void pullValue(int* table, int* val, size_t idx) {
  if(blockIdx.x * blockDim.x * threadIdx.x == 0) {
    *val = table[idx];
  }
}

int knapsack(std::vector<Stock>& stocks, size_t num_items, int budget) {

  int* val;
  unsigned short* chosen;
  int* table;
  hipMallocManaged(&val, 1 * sizeof(int));
  *val = 0;

  int* item_weights;
  int* item_values;
  int* item_quantities;

  allocateItems(stocks, item_weights, item_values, item_quantities, chosen);
  allocateTable(table, num_items, budget);
  void* args[] = {
    &item_weights,
    &item_values,
    &item_quantities,
    &chosen,
    &table,
    &num_items,
    &budget
  };
  for(int i = 1; i < 8192; ++i) {
    if(budget < (i*1024)) {
      dim3 dimGrid(i, 1, 1);
      printf("Launching %d blocks\n", i);
      dim3 dimBlock(1024, 1, 1);
      hipLaunchCooperativeKernel((void*)knapsackKernel, dimGrid, dimBlock, args);
      hipDeviceSynchronize();
      // knapsackKernel<<< i, 1024 >>>(item_weights, item_values, item_quantities, chosen, table, num_items, budget);
      break;
    }
  }

  pullValue<<< 1, 1 >>>(table, val, num_items*(budget+1) + budget);

  hipDeviceSynchronize();
  int v = *val;

  freeItems(item_weights, item_values, item_quantities, chosen);
  freeTable(table);
  hipFree(val);

  return v;
}

void generate_test(std::vector<Stock>& stocks) {
  stocks.push_back(Stock(0, 10, 5, std::vector<std::pair<int, int>>(1, {100, 60})));
  stocks.push_back(Stock(0, 20, 4, std::vector<std::pair<int, int>>(1, {100, 100})));
  stocks.push_back(Stock(0, 30, 2, std::vector<std::pair<int, int>>(1, {100, 120})));
  srand(time(NULL));
  for(int i = 0; i < 0; ++i) {
    int w = rand() % 71 + 70;
    int v = rand() % 24 - 9;
    int q = rand() % 340;
    if(q) {
      // printf("%d %d %d\n", w, v, q);
    }
    stocks.push_back(Stock(0, w, q, std::vector<std::pair<int, int>>(1, {100, v})));
  }
}

int main() {

  std::vector<Stock> stocks;
  generate_test(stocks);
  int v = knapsack(stocks, stocks.size(), 165000);
  std::cout << v << std::endl;

  return 0;
}